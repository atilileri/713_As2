#include "hip/hip_runtime.h"
/*
* 713_Assignment 2
* In the assignment, I will implement a Automatic Contrast Enhancement algorithm on CPU.
*
* Algortihm and strategies are my own.
* This file contains the CPU version of the algorithm.
*/

#include <iostream>
#include <fstream>
#include <sstream>
#include "npp.h"
#include <windows.h>

//global variables for and function declerations for performance measurements
double PCFreq = 0.0;
__int64 CounterStart = 0;
void StartCounter();
double GetCounter();

// Function declarations.
Npp8u *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray);

void
WritePGM(char * sFileName, Npp8u * pDst_Host, int nWidth, int nHeight, int nMaxGray);

void
MinMax8uCPU(Npp8u * pSrc_Host, NppiSize oROI, Npp8u & nMin_Host, Npp8u & nMax_Host);

void
SubMin8uCPU(Npp8u * pDst_Host, Npp8u * pSrc_Host, NppiSize oROI, Npp8u nMin_Host);

void
MulDiv8uCPU(Npp8u * pDst_Host, NppiSize oROI, Npp8u nConstant, int nScaleFactorMinus1);


// Main function.
int
main(int argc, char ** argv)
{
	// Parameter declarations.
	// Since this is the CPU version, I only kept host parameters.
	// I did not change variable names for easier comparison.
	Npp8u * pSrc_Host, *pDst_Host;
	int   nMaxGray;
	Npp8u    nMin_Host=0, nMax_Host=0;
	NppiSize oROI;

	std::cout << "####### CPU VERSION #######" << std::endl;
	
	// Load image to the host.
	std::cout << "Load PGM file." << std::endl;
	pSrc_Host = LoadPGM("..\\input\\lena_before.pgm", oROI.width, oROI.height, nMaxGray);
	pDst_Host = new Npp8u[oROI.width * oROI.height];

	std::cout << "Process the image on CPU." << std::endl;

	//start counter for performance mesaurements
	StartCounter();

	// Compute the min and the max.
	MinMax8uCPU(pSrc_Host, oROI, nMin_Host, nMax_Host);

	// Subtract Min
	SubMin8uCPU(pDst_Host, pSrc_Host, oROI, nMin_Host);

	// Compute the optimal nConstant and nScaleFactor for integer operation see GTC 2013 Lab NPP.pptx for explanation
	// I will prefer integer arithmetic, Instead of using 255.0f / (nMax_Host - nMin_Host) directly
	int nScaleFactor = 0;
	int nPower = 1;
	while (nPower * 255.0f / (nMax_Host - nMin_Host) < 255.0f)
	{
		nScaleFactor++;
		nPower *= 2;
	}
	Npp8u nConstant = static_cast<Npp8u>(255.0f / (nMax_Host - nMin_Host) * (nPower / 2));

	// multiply by nConstant and divide by 2 ^ nScaleFactor-1
	MulDiv8uCPU(pDst_Host, oROI, nConstant, nScaleFactor - 1);
	
	std::cout << "Duration of CPU Run: " << GetCounter() << " microseconds" << std::endl;

	std::cout << "Work done!" << std::endl;

	// Output the result image.
	std::cout << "Output the PGM file." << std::endl;
	WritePGM("..\\output\\lena_after_CPU.pgm", pDst_Host, oROI.width, oROI.height, nMaxGray);

	// Clean up.
	std::cout << "Clean up." << std::endl;
	delete[] pSrc_Host;
	delete[] pDst_Host;

	return 0;
}

// Disable reporting warnings on functions that were marked with deprecated.
#pragma warning( disable : 4996 )

// Load PGM file.
Npp8u *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray)
{
	char aLine[256];
	FILE * fInput = fopen(sFileName, "r");
	if (fInput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	// First line: version
	fgets(aLine, 256, fInput);
	std::cout << "\tVersion: " << aLine;
	// Second line: comment
	fgets(aLine, 256, fInput);
	std::cout << "\tComment: " << aLine;
	fseek(fInput, -1, SEEK_CUR);
	// Third line: size
	fscanf(fInput, "%d", &nWidth);
	std::cout << "\tWidth: " << nWidth;
	fscanf(fInput, "%d", &nHeight);
	std::cout << " Height: " << nHeight << std::endl;
	// Fourth line: max value
	fscanf(fInput, "%d", &nMaxGray);
	std::cout << "\tMax value: " << nMaxGray << std::endl;
	while (getc(fInput) != '\n');
	// Following lines: data
	Npp8u * pSrc_Host = new Npp8u[nWidth * nHeight];
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			pSrc_Host[i*nWidth + j] = fgetc(fInput);
	fclose(fInput);

	return pSrc_Host;
}

// Write PGM image.
void
WritePGM(char * sFileName, Npp8u * pDst_Host, int nWidth, int nHeight, int nMaxGray)
{
	FILE * fOutput = fopen(sFileName, "w+");
	if (fOutput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	char * aComment = "# Created by NPP";
	fprintf(fOutput, "P5\n%s\n%d %d\n%d\n", aComment, nWidth, nHeight, nMaxGray);
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			fputc(pDst_Host[i*nWidth + j], fOutput);
	fclose(fOutput);
}

// Calculate Min and Max
void
MinMax8uCPU(Npp8u * pSrc_Host, NppiSize oROI, Npp8u & nMin_Host, Npp8u & nMax_Host)
{
	nMin_Host = nMax_Host = pSrc_Host[0];

	for (Npp16u i = 0; i < oROI.height; i++)
	{
		for (Npp16u j = 0; j < oROI.width; j++)
		{
			if (nMin_Host > pSrc_Host[i * oROI.width + j])
			{
				nMin_Host = pSrc_Host[i * oROI.width + j];
			}
			else if (nMax_Host < pSrc_Host[i * oROI.width + j])
			{
				nMax_Host = pSrc_Host[i * oROI.width + j];
			}
		}
	}
}

// Subtract Min from Source and set it to Destination
void
SubMin8uCPU(Npp8u * pDst_Host, Npp8u * pSrc_Host, NppiSize oROI, Npp8u nMin_Host)
{
	for (Npp16u i = 0; i < oROI.height; i++)
	{
		for (Npp16u j = 0; j < oROI.width; j++)
		{
			pDst_Host[i * oROI.width + j] = pSrc_Host[i * oROI.width + j] - nMin_Host;
		}
	}
}

// multiply by nConstant and divide by 2 ^ nScaleFactor-1
void
MulDiv8uCPU(Npp8u * pDst_Host, NppiSize oROI, Npp8u nConstant, int nScaleFactorMinus1)
{
	for (Npp16u i = 0; i < oROI.height; i++)
	{
		for (Npp16u j = 0; j < oROI.width; j++)
		{
			pDst_Host[i * oROI.width + j] = static_cast<Npp8u>(round(pDst_Host[i * oROI.width + j] * nConstant / pow(2,nScaleFactorMinus1)));
		}
	}
}

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		std::cout << "QueryPerformanceFrequency failed!\n";

	PCFreq = double(li.QuadPart) / 1000000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}
