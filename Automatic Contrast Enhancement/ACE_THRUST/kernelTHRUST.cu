#include "hip/hip_runtime.h"
/*
* 713_Assignment 2
* In the assignment, I will implement a Automatic Contrast Enhancement algorithm with Parallel Reduction on CUDA.
*
* Algortihm and strategies are my own.
* This file contains the CUDA version of the algorithm.
*/

#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "npp.h"
#include <windows.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>

//global variables for and function declerations for performance measurements
double PCFreq = 0.0;
__int64 CounterStart = 0;
void StartCounter();
double GetCounter();

// Function declarations.
thrust::host_vector<Npp8u>
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray);

void
WritePGM(char * sFileName, thrust::host_vector<Npp8u> pDst_Host, int nWidth, int nHeight, int nMaxGray);

// Main function.
int
main(int argc, char ** argv)
{
	// Host parameter declarations.	
	int   nWidth, nHeight, nMaxGray;

	std::cout << "####### THRUST VERSION #######" << std::endl;

	// Load image to the host.
	std::cout << "Load PGM file." << std::endl;
	thrust::host_vector<Npp8u> vecHost = LoadPGM("..\\input\\lena_before.pgm", nWidth, nHeight, nMaxGray);
	// Device parameter declarations.
	Npp8u nMin, nMax;

	// Copy the image from the host to GPU
	thrust::device_vector<Npp8u> vecDev = vecHost;
	std::cout << "Copy image from host to device." << std::endl;
	std::cout << "Process the image on GPU." << std::endl;

	//start counter for performance mesaurements
	StartCounter();

	// Compute the min and the max.
	nMin = thrust::reduce(vecDev.begin(), vecDev.end(), nMaxGray, thrust::minimum<int>());
	nMax = thrust::reduce(vecDev.begin(), vecDev.end(), 0, thrust::maximum<int>());

	std::cout << "Duration after MinMax: " << GetCounter() << " microseconds" << std::endl;

	// Compute the optimal nConstant and nScaleFactor for integer operation see GTC 2013 Lab NPP.pptx for explanation
	// I will prefer integer arithmetic, Instead of using 255.0f / (nMax - nMin) directly
	int nScaleFactor = 0;
	int nPower = 1;
	while (nPower * 255.0f / (nMax - nMin) < 255.0f)
	{
		nScaleFactor++;
		nPower *= 2;
	}
	float nConstant = 255.0f / (nMax - nMin) * (nPower / 2);

	// Calculate nMultiplier by multiplying nConstant and divide by divider = 2 ^ (nScaleFactor-1)
	int nDivider = 1;
	for (int j = 0; j < nScaleFactor - 1; j++) nDivider <<= 1;

	float nMultiplier = nConstant / nDivider;
	
	// Subtract nMin and multiply by nMultiplier
	thrust::for_each(vecDev.begin(), vecDev.end(), thrust::placeholders::_1 = (thrust::placeholders::_1 - nMin) * nMultiplier);
		
	std::cout << "Duration of THRUST Run: " << GetCounter() << " microseconds" << std::endl;

	// Copy result back to the host.
	std::cout << "Work done! Copy the result back to host." << std::endl;
	vecHost = vecDev;

	// Output the result image.
	std::cout << "Output the PGM file." << std::endl;
	WritePGM("..\\output\\lena_after_THRUST.pgm", vecHost, nWidth, nHeight, nMaxGray);

	return 0;
}

// Disable reporting warnings on functions that were marked with deprecated.
#pragma warning( disable : 4996 )

// Load PGM file.
thrust::host_vector<Npp8u>
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray)
{
	char aLine[256];
	FILE * fInput = fopen(sFileName, "r");
	if (fInput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	// First line: version
	fgets(aLine, 256, fInput);
	std::cout << "\tVersion: " << aLine;
	// Second line: comment
	fgets(aLine, 256, fInput);
	std::cout << "\tComment: " << aLine;
	fseek(fInput, -1, SEEK_CUR);
	// Third line: size
	fscanf(fInput, "%d", &nWidth);
	std::cout << "\tWidth: " << nWidth;
	fscanf(fInput, "%d", &nHeight);
	std::cout << " Height: " << nHeight << std::endl;
	// Fourth line: max value
	fscanf(fInput, "%d", &nMaxGray);
	std::cout << "\tMax value: " << nMaxGray << std::endl;
	while (getc(fInput) != '\n');
	// Following lines: data
	thrust::host_vector<Npp8u> vecHost(nWidth * nHeight);
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			vecHost[i*nWidth + j] = fgetc(fInput);
	fclose(fInput);

	return vecHost;
}

// Write PGM image.
void
WritePGM(char * sFileName, thrust::host_vector<Npp8u> vecHost, int nWidth, int nHeight, int nMaxGray)
{
	FILE * fOutput = fopen(sFileName, "wb");
	if (fOutput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	char * aComment = "# Created by NPP";
	fprintf(fOutput, "P5\n%s\n%d %d\n%d\n", aComment, nWidth, nHeight, nMaxGray);
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			fputc(vecHost[i*nWidth + j], fOutput);
	fclose(fOutput);
}

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		std::cout << "QueryPerformanceFrequency failed!\n";

	PCFreq = double(li.QuadPart) / 1000000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}
