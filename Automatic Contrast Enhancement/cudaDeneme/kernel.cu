#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "npp.h"



// Main function.
int
main(int argc, char ** argv)
{
	// Host parameter declarations.	
	Npp8u * pSrc_Host, *pDst_Host;
	int   nWidth, nHeight, nMaxGray;

	std::cout << "####### CUDA VERSION #######" << std::endl;

	// Load image to the host.
	std::cout << "Load PGM file." << std::endl;
	//pSrc_Host = LoadPGM("..\\input\\lena_before.pgm", nWidth, nHeight, nMaxGray);
	pSrc_Host = new Npp8u[nWidth * nHeight];
	pDst_Host = new Npp8u[nWidth * nHeight];

	// Device parameter declarations.
	Npp8u	 * pSrc_Dev, *pDst_Dev;
	Npp8u    * pMin_Dev, *pMax_Dev;
	Npp8u    * pBuffer_Dev;
	Npp8u    nMin_Host, nMax_Host;
	NppiSize oROI;
	int		 nSrcStep_Dev, nDstStep_Dev;
	int		 nBufferSize_Host = 0;

	// Copy the image from the host to GPU
	oROI.width = nWidth;
	oROI.height = nHeight;
	pSrc_Dev = nppiMalloc_8u_C1(nWidth, nHeight, &nSrcStep_Dev);
	pDst_Dev = nppiMalloc_8u_C1(nWidth, nHeight, &nDstStep_Dev);
	std::cout << "Copy image from host to device." << std::endl;
	hipMemcpy2D(pSrc_Dev, nSrcStep_Dev, pSrc_Host, nWidth, nWidth, nHeight, hipMemcpyHostToDevice);

	std::cout << "Process the image on GPU." << std::endl;
	// Allocate device buffer for the MinMax primitive -- this is only necessary for nppi, we can simply return into nMin_Host and n_Max_Host
	hipMalloc(reinterpret_cast<void **>(&pMin_Dev), sizeof(Npp8u)); // You won't need these lines
	hipMalloc(reinterpret_cast<void **>(&pMax_Dev), sizeof(Npp8u)); // You won't need these lines
	nppiMinMaxGetBufferHostSize_8u_C1R(oROI, &nBufferSize_Host);  // You won't need these lines 
	hipMalloc(reinterpret_cast<void **>(&pBuffer_Dev), nBufferSize_Host); // You won't need these lines

																		   // REPLACE THIS PART WITH YOUR KERNELs
																		   // Compute the min and the max.
	nppiMinMax_8u_C1R(pSrc_Dev, nSrcStep_Dev, oROI, pMin_Dev, pMax_Dev, pBuffer_Dev); // // Replace this line with your KERNEL1 call (KERNEL1: your kernel calculating the minimum and maximum values and returning them here)
	hipMemcpy(&nMin_Host, pMin_Dev, sizeof(Npp8u), hipMemcpyDeviceToHost); // You won't need these lines to get the min and max. Return nMin_Host from your kernel function 
	hipMemcpy(&nMax_Host, pMax_Dev, sizeof(Npp8u), hipMemcpyDeviceToHost); // You won't need these lines to get the min and max. Return nMax_Host from your kernel function

	std::cout << "Min: " << static_cast<unsigned int>(nMin_Host) << " Max : " << static_cast<unsigned int>(nMax_Host) << std::endl;

	// Call SubC primitive.
	nppiSubC_8u_C1RSfs(pSrc_Dev, nSrcStep_Dev, nMin_Host, pDst_Dev, nDstStep_Dev, oROI, 0); // Replace this line with your KERNEL2 call (KERNEL2: your kernel subtracting the nMin_Host from all the pixels)

																							// Compute the optimal nConstant and nScaleFactor for integer operation see GTC 2013 Lab NPP.pptx for explanation
	int nScaleFactor = 0;
	int nPower = 1;
	while (nPower * 255.0f / (nMax_Host - nMin_Host) < 255.0f)
	{
		nScaleFactor++;
		nPower *= 2;
	}
	Npp8u nConstant = static_cast<Npp8u>(255.0f / (nMax_Host - nMin_Host) * (nPower / 2)); //you won't need these calculations

																						   // Call MulC primitive.
	nppiMulC_8u_C1IRSfs(nConstant, pDst_Dev, nDstStep_Dev, oROI, nScaleFactor - 1); // Replace this line with your KERNEL3 call (KERNEL3: your kernel multiplying all the pixels with the nConstant and then dividing them by nScaleFactor -1 to achieve: 255/(nMax_Host-nMinHost)))


																					//-------------------
																					// Copy result back to the host.
	std::cout << "Work done! Copy the result back to host." << std::endl;
	hipMemcpy2D(pDst_Host, nWidth * sizeof(Npp8u), pDst_Dev, nDstStep_Dev, nWidth * sizeof(Npp8u), nHeight, hipMemcpyDeviceToHost);

	// Output the result image.
	std::cout << "Output the PGM file." << std::endl;
	//WritePGM("..\\output\\lena_after_CUDA.pgm", pDst_Host, nWidth, nHeight, nMaxGray);

	// Clean up.
	std::cout << "Clean up." << std::endl;
	delete[] pSrc_Host;
	delete[] pDst_Host;

	nppiFree(pSrc_Dev);
	nppiFree(pDst_Dev);
	hipFree(pBuffer_Dev);
	nppiFree(pMin_Dev);
	nppiFree(pMax_Dev);

	return 0;
}



//int *g_idata, int *g_odata
__global__ void MinMax8uGPU(Npp8u * pSrc_Host, NppiSize oROI, Npp8u & nMin_Host, Npp8u & nMax_Host)
{
	extern __shared__ Npp8u sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = pSrc_Host[i];


	//// Each thread calculates C[row][col]
	//int row = blockIdx.y * blockDim.y + threadIdx.y;
	//int col = blockIdx.x * blockDim.x + threadIdx.x;
	//int temp = 0;
	//// Return if size is reached
	//if (row >= M || col >= P) return;
	////multiply every element and add to a temporary variable
	//for (int i = 0; i < N; i++)
	//{
	//	temp += A[(row * N) + i] * B[col + (i * P)];
	//}
	//C[(row * P) + col] = temp;
}
